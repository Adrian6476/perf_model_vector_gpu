#include "hip/hip_runtime.h"
#include <algorithm>
#include <fstream>
#include <iostream>
#include <iomanip>
#include "tools.cuh"

__global__ void function_a_kernel(const double* A, const double* x, double* y, const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        y[i] = 0;
        for (int j = 0; j < N; j++) {
            y[i] += A[i * N + j] * x[i];
        }
    }
}

__global__ void function_b_kernel(const double a, const double* u, const double* v, double* x, const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        x[i] = a * u[i] + v[i];
    }
}

__global__ void function_c_kernel(const double s, const double* x, const double* y, double* z, const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        if (i % 2 == 0) {
            z[i] = s * x[i] + y[i];
        } else {
            z[i] = x[i] + y[i];
        }
    }
}

__global__ void function_d_kernel(const double* u, const double* v, double* s, const int N) {
    __shared__ double shared_s;
    if (threadIdx.x == 0) {
        shared_s = 0;
    }
    __syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        atomicAdd(&shared_s, u[i] * v[i]);
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        atomicAdd(s, shared_s);
    }
}

void init_datastructures(double* u, double* v, double* A, const int N) {
    for (unsigned int i = 0; i < N; i++) {
        u[i] = static_cast<double>(i % 2);
        v[i] = static_cast<double>(i % 4);
    }
    for (unsigned int i = 0; i < N * N; i++) {
        A[i] = static_cast<double>(i % 8);
    }
}

void print_results_to_file(const double s, const double* x, const double* y,
                           const double* z, const double* A, const long long n,
                           std::ofstream& File) {
    unsigned int N = std::min(n, static_cast<long long>(30));

    File << "N: "
         << "\n"
         << n << "\n";
    File << "s: "
         << std::fixed
         << std::setprecision(1)
         << "\n"
         << s << "\n";
    File << "x: "
         << "\n";
    for (unsigned int i = 0; i < N; i++) {
        File << x[i] << " ";
    }
    File << "\n";
    File << "y: "
         << "\n";
    for (unsigned int i = 0; i < N; i++) {
        File << y[i] << " ";
    }
    File << "\n";
    File << "z: "
         << "\n";
    for (unsigned int i = 0; i < N; i++) {
        File << z[i] << " ";
    }
    File << "\n";
}

int main(int argc, char** argv) {
    long long N;
    if (argc == 2) {
        N = std::stoi(argv[1]);
    } else {
        std::cout << "Error: Missing problem size N. Please provide N as "
                     "commandline parameter. Usage example for N=10: "
                     "./number_crunching 10"
                  << std::endl;
        exit(0);
    }

    // Set up GPU
    setGPU();

    // Allocate memory on host
    double* u = new double[N];
    double* v = new double[N];
    double* A = new double[N * N];
    double* x = new double[N];
    double* y = new double[N];
    double* z = new double[N];

    // Initialize data on host
    init_datastructures(u, v, A, N);

    // Allocate memory on device
    double* d_u;
    double* d_v;
    double* d_A;
    double* d_x;
    double* d_y;
    double* d_z;
    double* d_s;
    ErrorCheck(hipMalloc((void**)&d_u, N * sizeof(double)), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((void**)&d_v, N * sizeof(double)), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((void**)&d_A, N * N * sizeof(double)), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((void**)&d_x, N * sizeof(double)), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((void**)&d_y, N * sizeof(double)), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((void**)&d_z, N * sizeof(double)), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((void**)&d_s, sizeof(double)), __FILE__, __LINE__);

    // Copy data from host to device
    ErrorCheck(hipMemcpy(d_u, u, N * sizeof(double), hipMemcpyHostToDevice), __FILE__, __LINE__);
    ErrorCheck(hipMemcpy(d_v, v, N * sizeof(double), hipMemcpyHostToDevice), __FILE__, __LINE__);
    ErrorCheck(hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice), __FILE__, __LINE__);

    // Define grid and block dimensions
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);

    // Launch kernels
    function_d_kernel<<<1, blockSize>>>(d_u, d_v, d_s, N);
    ErrorCheck(hipGetLastError(), __FILE__, __LINE__);
    ErrorCheck(hipDeviceSynchronize(), __FILE__, __LINE__);

    function_b_kernel<<<gridSize, blockSize>>>(2, d_u, d_v, d_x, N);
    ErrorCheck(hipGetLastError(), __FILE__, __LINE__);
    ErrorCheck(hipDeviceSynchronize(), __FILE__, __LINE__);

    function_a_kernel<<<gridSize, blockSize>>>(d_A, d_x, d_y, N);
    ErrorCheck(hipGetLastError(), __FILE__, __LINE__);
    ErrorCheck(hipDeviceSynchronize(), __FILE__, __LINE__);

    function_c_kernel<<<gridSize, blockSize>>>(0, d_x, d_y, d_z, N);
    ErrorCheck(hipGetLastError(), __FILE__, __LINE__);
    ErrorCheck(hipDeviceSynchronize(), __FILE__, __LINE__);

    // Copy results from device to host
    double s;
    ErrorCheck(hipMemcpy(&s, d_s, sizeof(double), hipMemcpyDeviceToHost), __FILE__, __LINE__);
    ErrorCheck(hipMemcpy(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost), __FILE__, __LINE__);
    ErrorCheck(hipMemcpy(y, d_y, N * sizeof(double), hipMemcpyDeviceToHost), __FILE__, __LINE__);
    ErrorCheck(hipMemcpy(z, d_z, N * sizeof(double), hipMemcpyDeviceToHost), __FILE__, __LINE__);

    // Print results to file
    std::ofstream File("partial_results.out");
    print_results_to_file(s, x, y, z, A, N, File);

    std::cout << "For correctness checking, partial results have been written to "
                 "partial_results.out"
              << std::endl;

    // Free memory on device
    ErrorCheck(hipFree(d_u), __FILE__, __LINE__);
    ErrorCheck(hipFree(d_v), __FILE__, __LINE__);
    ErrorCheck(hipFree(d_A), __FILE__, __LINE__);
    ErrorCheck(hipFree(d_x), __FILE__, __LINE__);
    ErrorCheck(hipFree(d_y), __FILE__, __LINE__);
    ErrorCheck(hipFree(d_z), __FILE__, __LINE__);
    ErrorCheck(hipFree(d_s), __FILE__, __LINE__);

    // Free memory on host
    delete[] u;
    delete[] v;
    delete[] A;
    delete[] x;
    delete[] y;
    delete[] z;

    return 0;
}
